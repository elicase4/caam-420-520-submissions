#include "hip/hip_runtime.h"
const int n_rows = 1280;
const int n_cols = 320;

const int N_rows = n_rows / 32;
const int N_cols = n_cols / 32;

const int N = 64;

void launch_gpu_matrix_scale(float* A_cpu, float* B_cpu, float* D_cpu) {
    // Pointers for memory on the GPU
    float* A_gpu = new float[N*N_cols]; 
    float* B_gpu = new float[N*N_cols]; 
    float* D_gpu = new float[N_cols]; 

    // Allocate memory on the GPU
    hipMalloc(&A_gpu, (N*N_cols)*sizeof(float));
    hipMalloc(&B_gpu, (N*N_cols)*sizeof(float));
    hipMalloc(&D_gpu, N_cols*sizeof(float));

    // Copy memory to the GPU
    hipMemcpy(A_gpu, A_cpu, (N*N_cols), cudaMemCopyHostToDeivce);
    hipMemcpy(B_gpu, B_cpu, (N*N_cols), cudaMemCopyHostToDeivce);
    hipMemcpy(D_gpu, D_cpu, N_cols, cudaMemCopyHostToDeivce);

    // Compute how much shared memory is needed
    int D_sharedMem = n_cols*sizeof(float);

    // Compute the size of the CUDA grid (i.e. the number of blocks in each dim)
    dim3 threadsPerBlock(N); 
    dim3 num_blocks(n_rows/N, N_cols);

    // Invoke the CUDA kernel
    matrix_scale_wrapped<<<num_blocks, threadsPerBlock, D_sharedMem>>>(A_gpu, B_gpu, D_gpu, n_rows, n_cols, N);

    // Copy memory back to the CPU
    hipMemcpy(A_gpu, A_cpu, (N*N_cols), cudaMemCopyDeviceToHost);
    hipMemcpy(B_gpu, B_cpu, (N*N_cols), cudaMemCopyDeviceToHost);
    hipMemcpy(D_gpu, D_cpu, N_cols, cudaMemCopyDeviceToHost);

    // Free memory on the GPU
    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(D_gpu);
}

__global__ matrix_scale_wrapped(float* A, float* B, float* D, const int n_rows, const int n_cols, const int N) {
    
    // Shared memory for D
    extern __shared__ float D_shared[];
   
    // Thread ID 
    int I = blockIdx.x;

    // Block ID
    int Dx = blockDim.x;
    int Dy = blockDim.y;

    // Perform Computation
    for (int i = 0; i < N; i++){
        for (int j = 0; j < N_cols; j++){
            if (i == j){
                B[i + j*N_cols] = A[i + j*N_cols]*D[j];
            } else {
                B[i + j*N_cols] = 0.0;
            }
        }
    }

}

int main() {
    // Pointers for the matrices/arrays
    float* A;
    float* B;
    float* D;

    // Allocate the arrays
    A = new float[n_rows*n_cols];
    B = new float[n_rows*n_cols];
    D = new float[n_cols];

    for (int i = 0; i < n_rows; i++) {
        for (int j = 0; j < n_cols; j++) {
            A[j + i*n_cols] = 1;
        }
    }

    for(int j = 0; j < n_cols; j++)
        D[j] = j;

    launch_gpu_matrix_scale(A, B, D);

    // Check B
    int correct = 0;
    for (int i = 0; i < n_rows; i++) {
        for (int j = 0; j < n_cols; j++) {
            if (B[j + i *n_cols] != j)
                correct = -1;
        }
    }

    delete[] A;
    delete[] B;
    delete[] D;
    
    return correct;
}
